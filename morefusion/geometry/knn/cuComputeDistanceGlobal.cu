
#include <hip/hip_runtime.h>
// #define IDX2D(i, j, dj) (dj * i + j)
// #define IDX3D(i, j, k, dj, dk) (IDX2D(IDX2D(i, j, dj), k, dk))

// #define BLOCK 512
// #define MAX_STREAMS 512
#define BLOCK_DIM 16


/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  *
  * @param A     pointer on the matrix A
  * @param wA    width of the matrix A = number of points in A
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */
extern "C" __global__
void cuComputeDistanceGlobal( float* A, int wA,
    float* B, int wB, int dim, float* AB){

// Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
__shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
__shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];


  // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
  __shared__ int begin_A;
  __shared__ int begin_B;
  __shared__ int step_A;
  __shared__ int step_B;
  __shared__ int end_A;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Other variables
  float tmp;
  float ssd = 0;

  // Loop parameters
  begin_A = BLOCK_DIM * blockIdx.y;
  begin_B = BLOCK_DIM * blockIdx.x;
  step_A  = BLOCK_DIM * wA;
  step_B  = BLOCK_DIM * wB;
  end_A   = begin_A + (dim-1) * wA;

    // Conditions
  int cond0 = (begin_A + tx < wA); // used to write in shared memory
  int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
  int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix

  // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
    // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
    if (a/wA + ty < dim){
      shared_A[ty][tx] = (cond0)? A[a + wA * ty + tx] : 0;
      shared_B[ty][tx] = (cond1)? B[b + wB * ty + tx] : 0;
    }
    else{
      shared_A[ty][tx] = 0;
      shared_B[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
    if (cond2 && cond1){
      for (int k = 0; k < BLOCK_DIM; ++k){
        tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp*tmp;
      }
    }

    // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory; each thread writes one element
  if (cond2 && cond1)
    AB[(begin_A + ty) * wB + begin_B + tx] = ssd;
}
